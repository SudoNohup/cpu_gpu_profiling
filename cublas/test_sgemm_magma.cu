#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include <hip/hip_runtime.h>

//#include <hip/hip_runtime_api.h>


#include "magma_gemm.cuh"

#define IDX2C(i, j, ld) (((j)*(ld)) + (i))
#define DATATYPE float
#define CUBLAS_FUNC hipblasSgemm 
//#define SIZE 1000

hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
	fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
	assert(result == hipSuccess);
  }
#endif
  return result;
}

static const char *cublasGetErrorString(hipblasStatus_t error) {
  switch (error)
  {
	case HIPBLAS_STATUS_SUCCESS:
	  return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
	  return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
	  return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
	  return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
	  return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
	  return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
	  return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
	  return "HIPBLAS_STATUS_INTERNAL_ERROR";
  }
  return "<unknown>";
}


hipblasStatus_t checkCublas(hipblasStatus_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != HIPBLAS_STATUS_SUCCESS) {
	fprintf(stderr, "CUBLAS Runtime Error: %s\n", cublasGetErrorString(result));
	assert(result == hipSuccess);
  }
#endif
  return result;
}

int main(int argc, char **argv) {
  int m, n, k;
  hipError_t cudaStat;
  hipblasStatus_t stat;
  int i, j;
  DATATYPE *A, *B, *C;
  int SIZE;


  int devId = 1;
  //if (argc > 1) devId = atoi(argv[1]);
  if (argc <=2 ) {
	printf("exactly 2 arguments\n");
	exit(0);
  }
  devId = atoi(argv[1]);
  SIZE = atoi(argv[2]);
  //printf("devId:%d, SIZE:%d\n", devId, SIZE);

  struct hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  //printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  m = n = k = SIZE;
  /*
  A = (DATATYPE *)malloc(m*k*sizeof(DATATYPE));
  B = (DATATYPE *)malloc(k*n*sizeof(DATATYPE));
  C = (DATATYPE *)malloc(m*n*sizeof(DATATYPE));
  */

  checkCuda( hipHostMalloc((void**)&A, m*k*sizeof(DATATYPE)) ); // host pinned
  checkCuda( hipHostMalloc((void**)&B, k*n*sizeof(DATATYPE)) ); // host pinned
  checkCuda( hipHostMalloc((void**)&C, m*n*sizeof(DATATYPE)) ); // host pinned

  for (j = 0; j < k; ++j) {
	for (i = 0; i < m; ++i) {
	  A[IDX2C(i, j, m)] = (DATATYPE)IDX2C(i, j, m);
	}
  }
  for (j = 0; j < n; ++j) {
	for (i = 0; i < k; ++i) {
	  B[IDX2C(i, j, m)] = 1.0;
	}
  }

  for (j = 0; j < n; ++j) {
	for (i = 0; i < m; ++i) {
	  C[IDX2C(i, j, m)] = 0.0;
	}
  }
  /*
  printf("A:\n");
  for (i = 0; i < m; ++i) {
	for (j = 0; j < k; ++j) {
	  printf("%5.0lf", A[IDX2C(i, j, m)]);
	}
	printf("\n");
  }
  printf("B:\n");
  for (i = 0; i < k; ++i) {
	for (j = 0; j < n; ++j) {
	  printf("%5.0lf", B[IDX2C(i, j, m)]);
	}
	printf("\n");
  }
  printf("C:\n");
  for (i = 0; i < m; ++i) {
	for (j = 0; j < n; ++j) {
	  printf("%5.0lf", C[IDX2C(i, j, m)]);
	}
	printf("\n");
  }
  */


  int ii;
  int nreps = 10;
  DATATYPE *d_A, *d_B, *d_C;
  //events for timing
  hipEvent_t start, stop, stop1, stop2, stop3;
  hipEvent_t execStart[nreps+1], execStop[nreps+1];
  checkCuda( hipEventCreate(&start) );
  checkCuda( hipEventCreate(&stop) );
  //checkCuda( hipEventCreate(&stop1) );
  //checkCuda( hipEventCreate(&stop2) );
  checkCuda( hipEventCreate(&stop3) );


  hipEventRecord(start,0);
  checkCuda(hipMalloc((void**)&d_A, m*k*sizeof(*A)));
  checkCuda(hipMalloc((void**)&d_B, m*k*sizeof(*B)));
  checkCuda(hipMalloc((void**)&d_C, m*k*sizeof(*C)));
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);

  float malloc_time;
  hipEventElapsedTime(&malloc_time, start, stop);
  //printf("Malloc time: %f\n", malloc_time);
  //printf("Malloc bandwidth\t: %f ms (%f GB/s)\n", malloc_time, (8*(m*k + k*n + m*n) * 1e-6)/ malloc_time);
  
  //printf("%f\t", (8*(m*k + k*n + m*n) * 1e-6)/ malloc_time);

  hipblasHandle_t handle;
  checkCublas(hipblasCreate(&handle));


  hipEventRecord(start,0);
  checkCuda(hipMemcpy(d_A, A, m*k*sizeof(*A), hipMemcpyHostToDevice));
  //hipEventRecord(stop1,0);
  checkCuda(hipMemcpy(d_B, B, k*n*sizeof(*B), hipMemcpyHostToDevice));
  //hipEventRecord(stop2,0);
  checkCuda(hipMemcpy(d_C, C, m*n*sizeof(*C), hipMemcpyHostToDevice));
  hipEventRecord(stop3,0);
  //checkCublas(hipblasSetMatrix(m, k, sizeof(*A), A, m, d_A, m));//A -> d_A
  //checkCublas(hipblasSetMatrix(k, n, sizeof(*B), B, k, d_B, k));//B -> d_B
  //checkCublas(hipblasSetMatrix(m, n, sizeof(*C), C, m, d_C, m));//C -> d_C
  //hipEventRecord(stop,0);
  //hipEventSynchronize(stop1);
  //hipEventSynchronize(stop2);
  hipEventSynchronize(stop3);

  float memcpy_h2d_time1, memcpy_h2d_time2, memcpy_h2d_time3;
  //hipEventElapsedTime(&memcpy_h2d_time1, start, stop1);
  //hipEventElapsedTime(&memcpy_h2d_time2, start, stop2);
  hipEventElapsedTime(&memcpy_h2d_time3, start, stop3);
  //printf("Memcpy_h2d_time: %f\n", memcpy_h2d_time);
  //printf(" Memcpy host to device\t: %f ms (%f GB/s)\n", memcpy_h2d_time, (8*(m*k + k*n + m*n) * 1e-6)/ memcpy_h2d_time);
  //printf("%f\t", (8*(m*k + k*n + m*n) * 1e-6)/ memcpy_h2d_time);
  //printf("%f\t", memcpy_h2d_time3);
  //printf("%f\t", memcpy_h2d_time2);
  //printf("%f\t", memcpy_h2d_time1);

  printf("Memcpy host to device\t: %f ms (%f GB/s)\n", memcpy_h2d_time3, (8*(m*k + k*n + m*n) * 1e-6)/ memcpy_h2d_time3);
  //printf"%f\t", (float)8.0 * (m*k + k*n + m*n) * 1e-6 / mem_h2d_time3


  //printf("0\t");

  DATATYPE alpha = 1.0;
  DATATYPE beta = 1.0;


  float cublas_time;


  //warm up

  hipEventRecord(start,0);
  ////checkCublas(CUBLAS_FUNC(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m));
  
  magma_sgemm(k, m, n, d_A, d_B, d_C);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&cublas_time, start, stop);
  //printf("cublas_time: %f\n", cublas_time);
  //printf(" Cublas\t: %f ms (%f GFLOPS)\n", cublas_time, (2.0*m*n*k* 1e-6)/ cublas_time);
  printf(" first-run Cublas GFLOPS\t: %f ms (%f GFLOPS)\n", cublas_time, (2.0*m*n*k* 1e-6)/ cublas_time);
  //printf("first-run GFLOPS:%f\n", (2.0*m*n*k* 1e-6)/ cublas_time);
  //printf("%f\t", cublas_time);


  float average_time, fastest_time = 10000000, slowest_time = 0, sum = 0.0;
  hipEventRecord(start,0);
  for (ii = 0; ii < nreps; ++ii) {
	//hipEventRecord(start,0);
	////checkCublas(CUBLAS_FUNC(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m));

	magma_sgemm(k, m, n, d_A, d_B, d_C);

	//hipEventRecord(stop,0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&cublas_time, start, stop);
	//printf("cublas_time: %f\n", cublas_time);
	//printf(" Cublas\t: %f ms (%f GFLOPS)\n", cublas_time, (2.0*m*n*k* 1e-6)/ cublas_time);
	//printf("%f\t", (2.0*m*n*k* 1e-6)/ cublas_time);
	
	//sum += cublas_time;
	//fastest_time = (fastest_time < cublas_time) ? fastest_time : cublas_time;
	//slowest_time = (slowest_time > cublas_time) ? slowest_time : cublas_time;
  }
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&cublas_time, start, stop);

  sum = cublas_time;

  average_time = sum/nreps;
  //printf("%f\t", (2.0*m*n*k* 1e-6)/ fastest_time);
  //printf("%f\t", (2.0*m*n*k* 1e-6)/ slowest_time);
  //printf("hot GFLOPS:%f\n", (2.0*m*n*k* 1e-6)/ average_time);
  printf(" hot-run Cublas GFLOPS\t: %f ms (%f GFLOPS)\n", average_time, (2.0*m*n*k* 1e-6)/ average_time);

  //printf("%f\t", fastest_time);
  //printf("%f\t", slowest_time);
  //printf("%f\t", average_time);


  hipEventRecord(start,0);
  checkCuda(hipMemcpy(C, d_C, m*n*sizeof(*C), hipMemcpyDeviceToHost));
  //checkCublas(hipblasGetMatrix(m, n, sizeof(*C), d_C, m, C, m));//d_C -> C
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float memcpy_d2h_time;
  hipEventElapsedTime(&memcpy_d2h_time, start, stop);
  //printf("Memcpy_d2h_time: %f\n", memcpy_d2h_time);
  //printf(" Memcpy device to host\t: %f ms (%f GB/s)\n", memcpy_d2h_time, (8*(m*n) * 1e-6)/ memcpy_d2h_time);
  //printf("%f\n", (8*(m*n) * 1e-6)/ memcpy_d2h_time);


  printf("Memcpy device to host\t: %f ms (%f GB/s)\n", memcpy_d2h_time, (8.0*(m*n) * 1e-6)/ memcpy_d2h_time);

  //printf("%f\t", (2.0*m*n*k* 1e-6)/ average_time);

  /*
  printf("C after dgemm:\n");
  for (i = 0; i < m; ++i) {
	for (j = 0; j < n; ++j) {
	  printf("%5.0lf", C[IDX2C(i, j, m)]);
	}
	printf("\n");
  }
  */


  printf("\n");

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipblasDestroy(handle);

  /*
  free(A);
  free(B);
  free(C);
  */
  hipHostFree(A);
  hipHostFree(B);
  hipHostFree(C);



  return EXIT_SUCCESS;
}
